#include "hip/hip_runtime.h"
/*
   Copyright 2016 Rafael Viana 20/08/18.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
 */

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"
#include "../include/cudaHeaders.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }


__device__ volatile int sem = 0;

__device__ void acquire_semaphore(volatile int *lock) {
    while (atomicCAS((int *) lock, 0, 1) != 0);
}

__device__ void release_semaphore(volatile int *lock) {
    *lock = 0;
    __threadfence();
}


inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__device__ int compare(char *String_1, char *String_2) {
    char TempChar_1,
            TempChar_2;

    do {
        TempChar_1 = *String_1++;
        TempChar_2 = *String_2++;
    } while (TempChar_1 && TempChar_1 == TempChar_2);

    return TempChar_1 - TempChar_2;
}

__device__ bool my_strcmp(char *array1, char *array2) {
    int i = 0;
    while (array1[i] != '\0') {
        if (array1[i] != array2[i]) {
            return false;
        }
        i++;
    }
    return true;
}

__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__device__ unsigned int count = 0;
__shared__ bool isLastBlockDone;
__device__ int index_elo_put;
__device__ unsigned int indexSetMap = 0;


__global__ void


frequencia_x2(__volatile__ EloVector *elo_k1, __volatile__ int elo_cur, Elo *set_elo, int *eloMapSizePointer,
              int minimo) {
    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x; //PC
    int eloMapSize = (*eloMapSizePointer);

    if (indexAtual < eloMapSize) {
        printf("Thread ");
        bool newFlag = true;
        int indexSetMap = 0;
        while (newFlag && indexSetMap < eloMapSize) {
            if ((0 == compare(elo_k1[elo_cur].eloArray[indexAtual].ItemId, set_elo[indexSetMap].ItemId)) &&
                (set_elo[indexSetMap].suporte >= minimo)) {
//                elo_new_put[0] = elo_k1[elo_k1_current].eloArray[indexAtual];
                printf("Thread %d Elo size %d AQUI %s %d\n",threadIdx.x,indexAtual, set_elo[indexAtual].ItemId,set_elo[indexAtual].suporte);
                newFlag = false;
            }
            indexSetMap++;
        }
    }
}

__global__ void frequencia_x1(Elo *set_elo, int *eloMapSizePointer, Elo *eloSetTemp, int *eloSetTempSize) {
    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x; //PC
    int eloMapSize = (*eloMapSizePointer);index_elo_put
    if (indexAtual < eloMapSize) {

        bool newFlag = true;

        int index = 0;

        while (newFlag && index < eloMapSize) {
            acquire_semaphore(&sem);
            if (0 == compare(eloSetTemp[index].ItemId, "")) {
//
                newFlag = false;
                (*eloSetTempSize) = (*eloSetTempSize) + 1;
                eloSetTemp[index] = set_elo[indexAtual];
//                printf("Sou Thread %d do Bloco %d Elo_k1 %s elox %s\n", indexAtual, blockIdx.x,
//                       elo_k1[elo_cur].eloArray[index].ItemId, set_elo[indexAtual].ItemId);

            } else if (0 == compare(eloSetTemp[index].ItemId,
                                    set_elo[indexAtual].ItemId)) {
                newFlag = false;
                eloSetTemp[index].suporte += set_elo[indexAtual].suporte;
//                printf("Sou Igual Thread %d do Bloco %d Elo_k1 %s elox %s\n", indexAtual, blockIdx.x,
//                       elo_k1[elo_cur].eloArray[index].ItemId, set_elo[indexAtual].ItemId);
            } else {
                index++;
            }
            release_semaphore(&sem);
        }

    }

}

__global__ void
pfp_growth(__volatile__ EloVector *elo_k1, __volatile__ int *elo_curr, ArrayMap *arrayMap, size_t arrayMapSize,
           Elo *elo_x, int *elo_int_x, int *minimo_suporte) {
    if (threadIdx.x == 0) {
        unsigned int value = atomicInc(&count, gridDim.x);
        isLastBlockDone = (value == (gridDim.x - 1));
    }

    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x;
    int elo_x_size = (*elo_int_x);
    if (indexAtual < elo_x_size) {
        int elo_cur = (*elo_curr);
        int xxx = 0;
        index_elo_put = 0;
        bool flag = true;
        Elo *Elo_k1 = (Elo *) malloc(sizeof(Elo) * elo_x_size);

        auto indexThreadArrayMap = elo_x[indexAtual].indexArrayMap;
        auto indexParentArrayMap = elo_x[indexAtual].indexArrayMap;
        while (flag) {
            indexParentArrayMap = arrayMap[indexParentArrayMap].indexP;
            if (arrayMap[indexThreadArrayMap].indexP != -1 &&
                arrayMap[indexParentArrayMap].indexP != -1) {
                my_cpcat(elo_x[indexAtual].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);

                Elo_k1[xxx].indexArrayMap = indexParentArrayMap;
                Elo_k1[xxx].suporte = elo_x[indexAtual].suporte;
            } else {
                flag = false;
            }
            xxx++;
        }
        int temp;
        for (int i = 0; i < (xxx - 1); ++i) {
            temp = atomicAdd(&index_elo_put, 1);
            elo_k1[elo_cur].eloArray[temp] = Elo_k1[i];
        }

        __syncthreads();

        if (isLastBlockDone) {
            if (temp == (index_elo_put - 1)) {
                (*elo_int_x) = temp;
                memset(elo_x, 0, sizeof(Elo) * index_elo_put);
                memcpy(elo_x, elo_k1[elo_cur].eloArray, sizeof(Elo) * index_elo_put);
                memset(elo_k1[elo_cur].eloArray, 0, sizeof(SetMap) * index_elo_put);

//     for (int i = 0; i < (*elo_int_x); ++i) {
//                    printf("%d CANDIDATO VAI PARA FREQUENCIA  Round :%d  | ELO :%s | IndexArray :%d | Suporte :%d\n",
//                           blockIdx.x,
//                           elo_cur, elo_x[i].ItemId, elo_x[i].indexArrayMap,
//                           elo_x[i].suporte);
//                }
                Elo *eloSetTemp = (Elo *) malloc(sizeof(Elo));
                eloSetTemp = (Elo *) malloc(sizeof(Elo) * index_elo_put);
                int *ss = (int *) malloc(sizeof(int));
                int block_size = 16;
                int blocks_per_row = (index_elo_put / block_size) + (index_elo_put % block_size > 0 ? 1 : 0);
                printf("Quantidade de Blocos %d Total %d\n", blocks_per_row, index_elo_put);
                frequencia_x1 << < blocks_per_row, block_size >> >
                                                   (elo_x, elo_int_x, eloSetTemp, ss);
                hipDeviceSynchronize();


                blocks_per_row = ( (*ss) / block_size) + ( (*ss) % block_size > 0 ? 1 : 0);

                frequencia_x2 << < blocks_per_row, block_size >> >
                                                   (elo_k1, elo_cur, eloSetTemp, ss, (*minimo_suporte));
                hipDeviceSynchronize();
                printf("AQUI DEPOIS SETMAP TESTE = %d\n", (*ss));
                for (int i = 0; i < (*ss); ++i) {
                    printf("VOLTA DA FREQUENCIA SEMAP  Round :%d  | ELO :%s | IndexArray :%d | Suporte :%d\n", elo_cur,
                           elo_k1[elo_cur].eloArray[i].ItemId, eloSetTemp[i].indexArrayMap,
                           elo_k1[elo_cur].eloArray[i].suporte);
                }
//                free(Elo_k1);
//                free(eloSetTemp->eloArray);
//                free(eloSetTemp);

            }
        }








//        if (temp == (index_elo_put - 1)) {
//
//            printf("Quem Sou eu ? %d bloco %d TesteTotal %d\n",threadIdx.x,blockIdx.x, atomicAdd(&teste, index_elo_put));
//
////            Elo *set_elo = (Elo *) malloc(sizeof(Elo) * index_elo_put);
////            memset(set_elo, 0, sizeof(Elo) * index_elo_put);
////            memset(elo_x,0,sizeof(Elo) * temp);
////            memcpy(elo_x,elo_k1[elo_cur].eloArray, sizeof(Elo) * temp);
//              int eloSize = 0;
////            for (int k = 0; k < index_elo_put; ++k) {
////
////      int i = 0;
////                bool flag = true;
////                while (i < index_elo_put && flag) {
////                    if (0 == compare(set_elo[i].ItemId, "")) {
////                        set_elo[i] = elo_k1[elo_cur].eloArray[k];
////                        eloSize++;
////                        flag = false;
////                    } else {
////                        if (0 == compare(elo_k1[elo_cur].eloArray[k].ItemId, set_elo[i].ItemId)) {
////                            flag = false;
////                            set_elo[i].suporte += elo_k1[elo_cur].eloArray[k].suporte;
////                        }
////                    }
////                    i++;
////                }
////            }
////

////
////            int x_threads = (*elo_int_x);
////            printf("AQUI CURR %d\n", (*elo_curr));
////            if(x_threads>0) {
////                *(elo_curr) = *(elo_curr) +1;
//////                printf("Chamando denovo com %d threads \n", x_threads);
////                pfp_growth << < 1, x_threads,x_threads*sizeof(Elo)*22>> >
////                                              (elo_k1, elo_curr, arrayMap, arrayMapSize,elo_xx,elo_int_x,minimo_suporte);
////////                hipDeviceSynchronize();
////            }
//////                free(elo_x);
////
//        }

    }
}


