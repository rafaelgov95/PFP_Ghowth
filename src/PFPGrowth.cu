//
// Created by rafael on 01/09/18.
//
#include "cudaHeaders.h"
#include "Kernel.h"
#include "PFPArray.h"
#include "PFPGrowth.cu.h"
#include "PFPArray.h"
#include "PFPArray.h"
#include "../include/PFPArray.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

PFPGrowth::PFPGrowth(ArrayMap *arrayMap, Elo *eloMap, size_t arrayMapSize, size_t eloPosMapSize) {
    ArrayMap *device_ArrayMap;
    Elo *device_elo_inicial;
    int *device_elosize_inical;
    int host_elosize_inical=(int)eloPosMapSize;

    EloVector *device_pointer_elo_vector, *host_elos_vector_and_memory_pointer_elos, *data_host_elos_vector;
    Elo *host_elos[eloPosMapSize];
    int *deviceEloVectorSize;
    int hostEloVectorSize=1;


    data_host_elos_vector = (EloVector *)malloc(sizeof(EloVector)*eloPosMapSize);
    for (int j = 0; j < eloPosMapSize; ++j) {
        data_host_elos_vector[j].eloArray=(Elo *)malloc(sizeof(Elo)*eloPosMapSize);
    }

    host_elos_vector_and_memory_pointer_elos = (EloVector*)malloc(eloPosMapSize * sizeof(EloVector));
    memcpy(host_elos_vector_and_memory_pointer_elos, data_host_elos_vector, eloPosMapSize * sizeof(EloVector));

    for (int i=0; i<eloPosMapSize; i++){
        hipMalloc(&(host_elos_vector_and_memory_pointer_elos[i].eloArray), eloPosMapSize*4*sizeof(Elo));
        hipMemcpy(host_elos_vector_and_memory_pointer_elos[i].eloArray, data_host_elos_vector[i].eloArray, eloPosMapSize*4*sizeof(Elo), hipMemcpyHostToDevice);
    }

    hipMalloc((void **)&device_pointer_elo_vector, sizeof(EloVector)*eloPosMapSize);
    hipMemcpy(device_pointer_elo_vector,host_elos_vector_and_memory_pointer_elos,sizeof(EloVector)*eloPosMapSize,hipMemcpyHostToDevice);

    gpuErrchk(hipMalloc((void **) &device_elo_inicial, sizeof(Elo) * eloPosMapSize*4));

    gpuErrchk(hipMalloc((void **) &device_ArrayMap, sizeof(ArrayMap) * arrayMapSize));
    gpuErrchk(hipMalloc((void **) &deviceEloVectorSize, sizeof(int)));

    gpuErrchk(hipMalloc((void **) &device_elosize_inical, sizeof(int)));

    gpuErrchk(hipMemcpy(device_ArrayMap, arrayMap, sizeof(ArrayMap) * arrayMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(device_elo_inicial, eloMap, sizeof(Elo) * eloPosMapSize, hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(device_elosize_inical, &host_elosize_inical, sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(deviceEloVectorSize,&hostEloVectorSize, sizeof(int), hipMemcpyHostToDevice));

    pfp_growth << < 1,eloPosMapSize,50*sizeof(Elo)>>>
                  (device_pointer_elo_vector,
                    deviceEloVectorSize,
                    device_ArrayMap,
                    arrayMapSize,
                          device_elo_inicial,
                          device_elosize_inical);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    for(int i =0;i<eloPosMapSize;++i) {
        host_elos[i] = (Elo *) malloc(eloPosMapSize*2* sizeof(Elo)); //Tamanho ficou pequeno para o final
    }

    gpuErrchk(hipMemcpy(host_elos_vector_and_memory_pointer_elos,device_pointer_elo_vector,sizeof(EloVector)*eloPosMapSize,hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&hostEloVectorSize,deviceEloVectorSize,sizeof(int),hipMemcpyDeviceToHost));


    for(int i =0;i<eloPosMapSize;++i){
        gpuErrchk(hipMemcpy(host_elos[i],host_elos_vector_and_memory_pointer_elos[i].eloArray,sizeof(Elo)*eloPosMapSize*2,hipMemcpyDeviceToHost)); //Tamanho ficou pequeno para o final

    }
    SetMap *setMap = (SetMap *)malloc(sizeof(SetMap)*eloPosMapSize);
    int intdex=0;
    for (int k = 0; k < eloPosMapSize; ++k) {
        strcmp(setMap[k].elo.ItemId, " ");
    }
    for (int k = 0; k < eloPosMapSize; ++k) {
        int i = 0;
        bool flag = true;
        while (i < eloPosMapSize && flag) {
            if (0 == strcmp(setMap[i].elo.ItemId,"")) {
                setMap[i].elo = eloMap[k];
                intdex++;
                flag = false;
            } else {
                if (0 == strcmp(eloMap[k].ItemId, setMap[i].elo.ItemId)) {
                    flag = false;
                    setMap[i].elo.suporte += eloMap[k].suporte;
                }
            }
            i++;
        }
    }
    for (int l = 0; l <intdex ; ++l) {
        host_elos[0][l]=setMap[l].elo;
    }
    host_elos_vector_and_memory_pointer_elos[0].size=intdex;


    printf("Total de Gerações de Frequência %d\n",hostEloVectorSize+1);
    for (int k = 0; k <=hostEloVectorSize+1; ++k) {
        for (int j = 0; j <host_elos_vector_and_memory_pointer_elos[k].size; ++j) {
            printf("%s;%d;%d \n",host_elos[k][j].ItemId,host_elos[k][j].indexArrayMap,host_elos[k][j].suporte);
        }
    }


    hipFree(device_ArrayMap);
    hipFree(deviceEloVectorSize);
    hipFree(device_pointer_elo_vector);
    hipFree(host_elos_vector_and_memory_pointer_elos->eloArray);
}
