#include "hip/hip_runtime.h"
//
// Created by rafael on 20/08/18.
//

#include <cudaHeaders.h>
#include "Kernel.h"
#include "PFPTree.h"
#include "PFPArray.h"
#include <hip/hip_runtime_api.h>
#include <cstdio>
#include "hip/hip_runtime.h"
#include "../include/PFPArray.h"


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}





__device__ int compare(char* String_1, char* String_2)
{
    char TempChar_1,
            TempChar_2;

    do
    {
        TempChar_1 = *String_1++;
        TempChar_2 = *String_2++;
    } while(TempChar_1 && TempChar_1 == TempChar_2);

    return TempChar_1 - TempChar_2;
}

__device__ bool my_strcmp( char *array1, char *array2) {
    int i = 0;
    while (array1[i] != '\0') {
        if (array1[i] != array2[i]) {
            return false;
        }
        i++;
    }
    return true;
}
__device__ char *my_strcpy(char *dest, const char *src) {
    int i = 0;
    do {
        dest[i] = src[i];
    } while (src[i++] != 0);
    return dest;
}

__device__ char *my_strcat(char *dest, const char *src) {
    int i = 0;
    while (dest[i] != 0) i++;
    my_strcpy(dest + i, src);
    return dest;
}

__device__ char *my_cpcat(const char *array1, const char *array2, char *src) {
    my_strcat(src, array1);
    my_strcat(src, array2);
    return src;
}

__device__ int index_elo_setmap;
__device__ int index_new_elo_setmap;
__device__ int index_elo_put;




__global__ void frequencia_x(EloVector *elo_k1,int elo_k1_current,Elo *elo_x,int *eloMapSizePointer, int minimo) {
    extern __shared__ SetMap setMap[];
    Elo *elo_new_put = (Elo*)malloc(sizeof((*eloMapSizePointer)));
    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x; //PC
    bool newFlag = true;
    int index_new_elo_setmap_local=0;
    int indexSetMap = 0;
    int eloSize = 0;
    int eloMapSize =(*eloMapSizePointer);
    memset(elo_new_put, 0, sizeof(Elo) * eloMapSize);
    memset(setMap, 0, sizeof(SetMap) * eloMapSize);
    index_elo_setmap =0;
    index_new_elo_setmap=0;
    index_new_elo_setmap_local=0;

    __syncthreads();
        if (threadIdx.x == 0) {
            for (int k = 0; k < eloMapSize; ++k) {

                my_strcpy(setMap[k].elo.ItemId, " ");
            }

        }
    __syncthreads();

        if (indexAtual == 0) {

            for (int k = 0; k < eloMapSize; ++k) {
                int i = 0;
                bool flag = true;
                while (i < eloMapSize && flag) {
                    if (0 == compare(setMap[i].elo.ItemId, " ")) {

                        setMap[i].elo = elo_x[k];

                        eloSize++;
                        flag = false;
                    } else {
                        if (0 == compare(elo_x[k].ItemId, setMap[i].elo.ItemId)) {
                            flag = false;
                            setMap[i].elo.suporte += elo_x[k].suporte;
                        }
                    }
                    i++;
                }

            }

            atomicAdd(&index_elo_setmap, eloSize-1);



        }
        __syncthreads();

        while (newFlag && indexSetMap <=index_elo_setmap) {
            if ((0 == compare(elo_x[indexAtual].ItemId, setMap[indexSetMap].elo.ItemId)) &&
                (setMap[indexSetMap].elo.suporte >= minimo)) {
                elo_new_put[index_new_elo_setmap_local] = elo_x[indexAtual];
//                printf("Thread %d Elo size %d AQUI %s %d\n",threadIdx.x,index_new_elo_setmap_local, elo_new_put[index_new_elo_setmap_local].ItemId,elo_new_put[index_new_elo_setmap_local].suporte);

                index_new_elo_setmap_local++;

                newFlag = false;
            }
            indexSetMap++;
        }
        __syncthreads();

    memset(elo_x, 0, sizeof(Elo) * eloMapSize);
    for (int l = 0; l <index_new_elo_setmap_local ; ++l) {
        elo_x[atomicAdd(&index_new_elo_setmap, 1)] = elo_new_put[l];
    }

    (*eloMapSizePointer)=index_new_elo_setmap;
    index_new_elo_setmap=0;
    __syncthreads();
           if (threadIdx.x <= index_elo_setmap) {
               if( (setMap[threadIdx.x].elo.suporte >= minimo)){
                    elo_k1[elo_k1_current].eloArray[atomicAdd(&index_new_elo_setmap, 1)] = setMap[threadIdx.x].elo;
                }
               elo_k1[elo_k1_current].size=index_new_elo_setmap;
            }
    __syncthreads();



}

__global__ void pfp_growth(EloVector *elo_k1, int *elo_curr ,ArrayMap *arrayMap,size_t arrayMapSize, Elo *elo_x, int *elo_int_x, int *minimo_suporte) {
    extern __shared__ Elo elo[];
    auto indexAtual = blockIdx.x * blockDim.x + threadIdx.x;

    int elo_cur= (*elo_curr);
    int elo_x_size= (*elo_int_x);
    int xxx = 0;
    bool flag = true;
    Elo *Elo_k1 = (Elo * ) malloc(sizeof(Elo) *elo_x_size);

    auto indexThreadArrayMap = elo_x[indexAtual].indexArrayMap;
    auto indexParentArrayMap = elo_x[indexAtual].indexArrayMap;
    while (flag) {
               indexParentArrayMap = arrayMap[indexParentArrayMap].indexP;
            if (arrayMap[indexThreadArrayMap].indexP != -1 &&
                arrayMap[indexParentArrayMap].indexP != -1) {
                my_cpcat(elo_x[indexAtual].ItemId,
                         arrayMap[indexParentArrayMap].ItemId, Elo_k1[xxx].ItemId);

                Elo_k1[xxx].indexArrayMap = indexParentArrayMap;
                Elo_k1[xxx].suporte = elo_x[indexAtual].suporte;
            } else {
                flag = false;
            }
            xxx++;
        }

// Algoritmo 1 End;
// Algoritmo 2 Begin;
    __syncthreads();
    memset(elo_x, 0, sizeof(Elo) * elo_x_size);

    for (int i = 0; i < (xxx - 1); ++i) {
            elo_x[atomicAdd(&index_elo_put, 1)] = Elo_k1[i];

        }

        if (threadIdx.x == elo_x_size - 1) {
            (*elo_int_x)=index_elo_put;
//            for (int i = 0; i < (*elo_int_x); ++i) {
//                printf("VAI DA MORTE  Round :%d  | ELO :%s | IndexArray :%d | Suporte :%d\n",elo_cur,elo_x[i].ItemId,elo_x[i].indexArrayMap,elo_x[i].suporte);
//            }
            frequencia_x << < 1, index_elo_put, sizeof(SetMap) * index_elo_put*2 >> >
                                                (elo_k1, elo_cur, elo_x,elo_int_x , (*minimo_suporte));

            hipDeviceSynchronize();
//            printf("AQUI DEPOIS %d\n", (*elo_int_x));
//            for (int i = 0; i < (*elo_int_x); ++i) {
//                printf("VOLTA DA MORTE  Round :%d  | ELO :%s | IndexArray :%d | Suporte :%d\n",elo_cur,elo_x[i].ItemId,elo_x[i].indexArrayMap,elo_x[i].suporte);
//            }
            index_elo_put = 0;
            if ( (*elo_int_x) > 0) {
                int x_threads = (*elo_int_x);
                *(elo_curr) = *(elo_curr) + 1;
//                printf("Chamando denovo com %d threads \n", x_threads);
                pfp_growth << < 1, x_threads, (*elo_int_x) * 4 * sizeof(Elo) >> >
                                              (elo_k1, elo_curr, arrayMap, arrayMapSize,elo_x,elo_int_x,minimo_suporte);
                hipPeekAtLastError();
                hipDeviceSynchronize();
            }
        }

}


